#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iomanip>
#include <fstream>
#include <sstream>
#include <iostream>
#include <cmath>
#include <time.h> 
#include <sys/time.h> 
#include <sys/resource.h> 
#include <string.h>
#include <map>
#include <cassert>
#include <vector>
#include <limits>

using namespace std;

#include "contraintes_gpu.h"

__global__
void slocal(int NB_SPH,R * LIST_IND, int * NBCONTCO, unsigned int * NOCONT, R * LIST_R, R * NCONT, R * FCJI, R * PSIG11, R * PSIG12, R * PSIG13, R * PSIG22, R * PSIG23, R * PSIG33, int NMAXZ) {

	int numcor = (blockIdx.x*blockDim.x)+threadIdx.x;

	if (numcor<NB_SPH) {

		R sig11=0.;
		R sig12=0.;
		R sig13=0.;
		R sig21=0.;
		R sig22=0.;
		R sig23=0.;
		R sig31=0.;
		R sig32=0.;
		R sig33=0.;  			   
		  
		R coefij=LIST_IND[numcor];

					 for(int kt=0;kt<NBCONTCO[numcor];kt++){ 

				                unsigned int numc=NOCONT[numcor*NMAXZ+kt]; 
//  if(numcor==0){ printf("NUMC: (%i,%i)\n",kt,numc);}
						R ray=LIST_R[numcor];
						R n1=-NCONT[numc*9+0];
						R n2=-NCONT[numc*9+1];
						R n3=-NCONT[numc*9+2];
/*
						sig11=numc;
						sig12=numc;
						sig13=numc;
						sig21=numc;
						sig22=numc;
						sig23=numc;
						sig31=numc;
						sig32=numc;
						sig33=numc;*/

						sig11=sig11+coefij*ray*n1*FCJI[numc*3+0];
						sig12=sig12+coefij*ray*n1*FCJI[numc*3+1];
						sig13=sig13+coefij*ray*n1*FCJI[numc*3+2];
						sig21=sig21+coefij*ray*n2*FCJI[numc*3+0];
						sig22=sig22+coefij*ray*n2*FCJI[numc*3+1];
						sig23=sig23+coefij*ray*n2*FCJI[numc*3+2];
						sig31=sig31+coefij*ray*n3*FCJI[numc*3+0];
						sig32=sig32+coefij*ray*n3*FCJI[numc*3+1];
						sig33=sig33+coefij*ray*n3*FCJI[numc*3+2];

					 }
	

				sig12=(sig12+sig21)/2.;
				sig13=(sig13+sig31)/2.;
				sig23=(sig23+sig32)/2.;                
						 
                               
				PSIG11[numcor]=sig11;
				PSIG12[numcor]=sig12;
				PSIG13[numcor]=sig13;                              
				PSIG22[numcor]=sig22;
				PSIG23[numcor]=sig23;                                      
				PSIG33[numcor]=sig33; 

	  	}
}

__global__
void shalo(int NB_SPH, int * NBHALO, unsigned int * NOHALO, R * VOLHALO, R * PSIG11, R * PSIG12, R * PSIG13, R * PSIG22, R * PSIG23, R * PSIG33, R * SIG11, R * SIG12, R * SIG13, R * SIG22, R * SIG23, R * SIG33, R * VONMIS, R * TRACE, R * SIG1, R *SIG2, R * SIG3, R minsig11, R maxsig11, R minsig12, R maxsig12, R minsig13, R maxsig13, R minsig22, R maxsig22, R minsig23, R maxsig23, R minsig33, R maxsig33, R minvm, R maxvm, R mintrac, R maxtrac, R minsig1, R maxsig1, R minsig2, R maxsig2, R minsig3, R maxsig3, int NMAXHALO) {
	
	int jt = (blockIdx.x*blockDim.x)+threadIdx.x;

	if (jt<NB_SPH) {

		R sig11=PSIG11[jt];
		R sig12=PSIG12[jt];
		R sig13=PSIG13[jt];
		R sig22=PSIG22[jt];
		R sig23=PSIG23[jt];
		R sig33=PSIG33[jt];


         	  for(int kt=1;kt<NBHALO[jt];kt++){ 
			//printf("NBHALOav: %i\n",long(700000)*NMAXHALO+kt);
			unsigned int lt=NOHALO[long(jt)*NMAXHALO+kt];			
			//printf("NBHALOap: %i\n",lt);
			sig11=sig11+PSIG11[lt];
			sig12=sig12+PSIG12[lt];
			sig13=sig13+PSIG13[lt];                             
			sig22=sig22+PSIG22[lt];
			sig23=sig23+PSIG23[lt];                                      
			sig33=sig33+PSIG33[lt];  
		  }


				sig11/=VOLHALO[jt];
				sig12/=VOLHALO[jt];
				sig13/=VOLHALO[jt];
				sig22/=VOLHALO[jt];
				sig23/=VOLHALO[jt];
				sig33/=VOLHALO[jt];	

				SIG11[jt]=sig11;
				SIG12[jt]=sig12;
				SIG13[jt]=sig13;
				SIG22[jt]=sig22;
				SIG23[jt]=sig23;
				SIG33[jt]=sig33;

				R i1=sig11+sig22+sig33;
				R i2=sig11*sig22+sig22*sig33+sig33*sig11-sig12*sig12-sig23*sig23-sig13*sig13;
				R i3=sig11*(sig22*sig33-sig23*sig23)-sig12*(sig12*sig33-sig13*sig23)+sig13*(sig12*sig23-sig22*sig13);
					
				R b=-i1;
				R c=i2;
				R d=-i3;
				R p=c-b*b/3.;
				R q=d-b*c/3.+2*b*b*b/27.;
				R detd=4*c*c*c+27*d*d+4*d*b*b*b-b*b*c*c-18*b*c*d; 


				R s1=0.;
				R s2=0.;
				R s3=0.;

				if (fabs(detd)<=1e-30){
				double t=-q/2.;
				s1=2.*pow(t,1./3)-b/3.;
				s2=-pow(t,1./3)-b/3.;
				s3=s2;  			
				}
				else{
				R r=sqrt(-p*p*p/27.);
				R theta=acos(-q/(2*r));
				s1=2.*sqrt(-p/3.)*cos(theta/3.)-b/3.;
				s2=2.*sqrt(-p/3.)*cos((theta+2.*3.14159265358979323846)/3.)-b/3.;
				s3=2.*sqrt(-p/3.)*cos((theta+4.*3.14159265358979323846)/3.)-b/3.;

				}
			    
			    
				R smax=max(max(s1,s2),s3);
				R smin=min(min(s1,s2),s3);
				if(s1==smax){s2=max(s2,s3);}
				else if(s2==smax){s2=max(s1,s3);}
				else if(s3==smax){s2=max(s1,s2);}              

				s1=smax;
				s3=smin;
				R trac=s1+s2+s3;                
				R vmis=sqrt((sig11-sig22)*(sig11-sig22)+(sig33-sig22)*(sig33-sig22)+(sig11-sig33)*(sig11-sig33)+6.*(sig12*sig12+sig13*sig13+sig23*sig23))/sqrt(2.);
				if(vmis!=vmis) {vmis=0.;}    

				VONMIS[jt]=vmis;
				TRACE[jt]=trac;
				SIG1[jt]=s1;
				SIG2[jt]=s2;
				SIG3[jt]=s3;
/*
				minvm=fmin(vmis,minvm);
				maxvm=fmax(vmis,maxvm);
				
				mintrac=fmin(trac,mintrac);
				maxtrac=fmax(trac,maxtrac);			
				
				minsig11=fmin(sig11,minsig11);
				maxsig11=fmax(sig11,maxsig11);
				minsig12=fmin(sig12,minsig12);
				maxsig12=fmax(sig12,maxsig12);
				minsig13=fmin(sig13,minsig13);
				maxsig13=fmax(sig13,maxsig13);				
				minsig22=fmin(sig22,minsig22);
				maxsig22=fmax(sig22,maxsig22);					
				minsig23=fmin(sig23,minsig23);
				maxsig23=fmax(sig23,maxsig23);	
				minsig33=fmin(sig33,minsig33);
				maxsig33=fmax(sig33,maxsig33);					
								
				minsig1=fmin(s1,minsig1);
				maxsig1=fmax(s1,maxsig1);
				minsig2=fmin(s2,minsig2);
				maxsig2=fmax(s2,maxsig2);	
				minsig3=fmin(s3,minsig3);
				maxsig3=fmax(s3,maxsig3);	*/
		}				
	}



void contrainteshalo_gpu(R Pi, R coef1, int ite,int NBENREG, int NB_SPH, int NBCO, int NMAXZ,int NMAXHALO, int NMAXCONT, R H_TOT, R V_TOT, R Z_TOT,R * LIST_R, R ** FCJI, R ** NCONT, unsigned int ** NOCONT, int * NBCONTCO, R * VONMIS, R * TRACE, R * SIG11, R * SIG12, R * SIG13, R * SIG22, R * SIG23, R * SIG33, R * SIG1, R * SIG2, R * SIG3,R &minvm, R &maxvm,R &mintrac, R &maxtrac, R &minsig11, R &maxsig11, R &minsig12, R &maxsig12, R &minsig13, R &maxsig13, R &minsig22, R &maxsig22, R &minsig23, R &maxsig23, R &minsig33, R &maxsig33, R &minsig1, R &maxsig1, R &minsig2, R &maxsig2, R &minsig3, R &maxsig3, bool * EDGE, unsigned int ** NOHALO, int * NBHALO, R * LIST_V, R * VOLHALO,R * LIST_IND) {

int it;

mintrac = 1e12;
maxtrac =  -1e12;
minvm = 1e12;
maxvm =  -1e12;
minsig11 = 1e12;
maxsig11 = -1e12;
minsig12 = 1e12;
maxsig12 = -1e12;
minsig13 = 1e12;
maxsig13 = -1e12;
minsig22 = 1e12;
maxsig22 = -1e12;
minsig23 = 1e12;
maxsig23 = -1e12;
minsig33 = 1e12;
maxsig33 = -1e12;
minsig1 = 1e12;
maxsig1 = -1e12;
minsig2 = 1e12;
maxsig2 = -1e12;
minsig3 = 1e12;
maxsig3 = -1e12;

/////////////////////////////////
// Vecteurs/matrices device
R * dLIST_IND;
R * dLIST_R;
R * dPSIG11;
R * dPSIG12;
R * dPSIG13;
R * dPSIG22;
R * dPSIG23;
R * dPSIG33;
R * dSIG11;
R * dSIG12;
R * dSIG13;
R * dSIG22;
R * dSIG23;
R * dSIG33;
R * dVONMIS;
R * dTRACE;
R * dSIG1;
R * dSIG2;
R * dSIG3;
R * dVOLHALO;
int * dNBHALO;
int * dNBCONTCO;
unsigned int * dNOCONT;
R * dNCONT;
R * dFCJI;
unsigned int * dNOHALO;

/////////////////////////////////
// Allocation mémoire

hipMalloc((void **)&dLIST_IND, NB_SPH*sizeof(R));
hipMemcpy(dLIST_IND, LIST_IND, NB_SPH*sizeof(R), hipMemcpyHostToDevice);

hipMalloc((void **)&dLIST_R, NB_SPH*sizeof(R));
hipMemcpy(dLIST_R, LIST_R, NB_SPH*sizeof(R), hipMemcpyHostToDevice);

hipMalloc((void **)&dPSIG11, NB_SPH*sizeof(R));
hipMalloc((void **)&dPSIG12, NB_SPH*sizeof(R));
hipMalloc((void **)&dPSIG13, NB_SPH*sizeof(R));
hipMalloc((void **)&dPSIG22, NB_SPH*sizeof(R));
hipMalloc((void **)&dPSIG23, NB_SPH*sizeof(R));
hipMalloc((void **)&dPSIG33, NB_SPH*sizeof(R));

hipMalloc((void **)&dNBCONTCO, NB_SPH*sizeof(int));
hipMemcpy(dNBCONTCO, NBCONTCO, NB_SPH*sizeof(int), hipMemcpyHostToDevice);

hipMalloc((void **)&dNOCONT, NB_SPH*NMAXZ*sizeof(unsigned int));
hipMemcpy(dNOCONT, NOCONT[0], NB_SPH*NMAXZ*sizeof(unsigned int), hipMemcpyHostToDevice);

hipMalloc((void **)&dNCONT, NMAXCONT*9*sizeof(R));
hipMemcpy(dNCONT, NCONT[0], NMAXCONT*9*sizeof(R), hipMemcpyHostToDevice);

hipMalloc((void **)&dFCJI, NMAXCONT*3*sizeof(R));
hipMemcpy(dFCJI, FCJI[0], NMAXCONT*3*sizeof(R), hipMemcpyHostToDevice);

/////////////////////////////////
// Contraintes à l'échelle de la particule

dim3 DimGrid ((NB_SPH-1)/256+1,1,1) ;
dim3 DimBlock (256,1,1) ;
slocal<<<DimGrid, DimBlock>>>(NB_SPH,dLIST_IND,dNBCONTCO,dNOCONT,dLIST_R,dNCONT,dFCJI,dPSIG11,dPSIG12, dPSIG13, dPSIG22,dPSIG23,dPSIG33,NMAXZ);
	
/////////////////////////////////
// Libération mémoire
	hipFree(dLIST_IND);
	hipFree(dLIST_R);
	hipFree(dNBCONTCO);
	hipFree(dNOCONT);
	hipFree(dNCONT);
	hipFree(dFCJI);

/////////////////////////////////
// Allocation mémoire

hipMalloc((void **)&dSIG11, NB_SPH*sizeof(R));
hipMalloc((void **)&dSIG12, NB_SPH*sizeof(R));
hipMalloc((void **)&dSIG13, NB_SPH*sizeof(R));
hipMalloc((void **)&dSIG22, NB_SPH*sizeof(R));
hipMalloc((void **)&dSIG23, NB_SPH*sizeof(R));
hipMalloc((void **)&dSIG33, NB_SPH*sizeof(R));
hipMalloc((void **)&dVONMIS, NB_SPH*sizeof(R));
hipMalloc((void **)&dTRACE, NB_SPH*sizeof(R));
hipMalloc((void **)&dSIG1, NB_SPH*sizeof(R));
hipMalloc((void **)&dSIG2, NB_SPH*sizeof(R));
hipMalloc((void **)&dSIG3, NB_SPH*sizeof(R));

hipMalloc((void **)&dVOLHALO, NB_SPH*sizeof(R));
hipMemcpy(dVOLHALO, VOLHALO, NB_SPH*sizeof(R), hipMemcpyHostToDevice);

hipMalloc((void **)&dNBHALO, NB_SPH*sizeof(int));
hipMemcpy(dNBHALO, NBHALO, NB_SPH*sizeof(int), hipMemcpyHostToDevice);

hipMalloc((void **)&dNOHALO, long(NB_SPH)*NMAXHALO*sizeof(unsigned int));
hipMemcpy(dNOHALO, NOHALO[0], long(NB_SPH)*NMAXHALO*sizeof(unsigned int), hipMemcpyHostToDevice);

size_t free, total;

printf("\n");

hipMemGetInfo(&free,&total);

printf("%d KB free of total %d KB\n",free/1024,total/1024);

/////////////////////////////////
// Contraintes à l'échelle du halo
shalo<<<DimGrid, DimBlock>>>(NB_SPH, dNBHALO,dNOHALO,dVOLHALO,dPSIG11,dPSIG12,dPSIG13,dPSIG22,dPSIG23,dPSIG33,dSIG11,dSIG12,dSIG13,dSIG22,dSIG23,dSIG33,dVONMIS,dTRACE,dSIG1,dSIG2,dSIG3,minsig11, maxsig11,minsig12,maxsig12,minsig13,maxsig13,minsig22,maxsig22,minsig23,maxsig23,minsig33,maxsig33,minvm,maxvm,mintrac,maxtrac,minsig1,maxsig1,minsig2,maxsig2,minsig3,maxsig3,NMAXHALO);

/////////////////////////////////
// Copies des vecteurs/matrices utiles vers l'host
    hipMemcpy(SIG11, dSIG11, NB_SPH*sizeof(R), hipMemcpyDeviceToHost);
    hipMemcpy(SIG12, dSIG12, NB_SPH*sizeof(R), hipMemcpyDeviceToHost);
    hipMemcpy(SIG13, dSIG13, NB_SPH*sizeof(R), hipMemcpyDeviceToHost);
    hipMemcpy(SIG22, dSIG22, NB_SPH*sizeof(R), hipMemcpyDeviceToHost);
    hipMemcpy(SIG23, dSIG23, NB_SPH*sizeof(R), hipMemcpyDeviceToHost);
    hipMemcpy(SIG33, dSIG33, NB_SPH*sizeof(R), hipMemcpyDeviceToHost);
    hipMemcpy(VONMIS, dVONMIS, NB_SPH*sizeof(R), hipMemcpyDeviceToHost);
    hipMemcpy(TRACE, dTRACE, NB_SPH*sizeof(R), hipMemcpyDeviceToHost);	
    hipMemcpy(SIG1, dSIG1, NB_SPH*sizeof(R), hipMemcpyDeviceToHost);
    hipMemcpy(SIG2, dSIG2, NB_SPH*sizeof(R), hipMemcpyDeviceToHost);
    hipMemcpy(SIG3, dSIG3, NB_SPH*sizeof(R), hipMemcpyDeviceToHost);

/////////////////////////////////
// Libération mémoire

	hipFree(dPSIG11);
	hipFree(dPSIG12);
	hipFree(dPSIG13);
	hipFree(dPSIG22);
	hipFree(dPSIG23);
	hipFree(dPSIG33);
	hipFree(dSIG11);
	hipFree(dSIG12);
	hipFree(dSIG13);
	hipFree(dSIG22);
	hipFree(dSIG23);
	hipFree(dSIG33);
	hipFree(dVONMIS);
	hipFree(dTRACE);
	hipFree(dSIG1);
	hipFree(dSIG2);
	hipFree(dSIG3);
	hipFree(dNBHALO);
	hipFree(dVOLHALO);
	hipFree(dNOHALO);

/////////////////////////////////
// Traitement Min/max


for(it=0;it<NB_SPH;it++){
				minvm=min(VONMIS[it],minvm);
				maxvm=max(VONMIS[it],maxvm);
				
				mintrac=min(TRACE[it],mintrac);
				maxtrac=max(TRACE[it],maxtrac);			
				
				minsig11=min(SIG11[it],minsig11);
				maxsig11=max(SIG11[it],maxsig11);
				minsig12=min(SIG12[it],minsig12);
				maxsig12=max(SIG12[it],maxsig12);
				minsig13=min(SIG13[it],minsig13);
				maxsig13=max(SIG13[it],maxsig13);				
				minsig22=min(SIG22[it],minsig22);
				maxsig22=max(SIG22[it],maxsig22);					
				minsig23=min(SIG23[it],minsig23);
				maxsig23=max(SIG23[it],maxsig23);	
				minsig33=min(SIG33[it],minsig33);
				maxsig33=max(SIG33[it],maxsig33);						
								
				minsig1=min(SIG1[it],minsig1);
				maxsig1=max(SIG1[it],maxsig1);
				minsig2=min(SIG2[it],minsig2);
				maxsig2=max(SIG2[it],maxsig2);	
				minsig3=min(SIG3[it],minsig3);
				maxsig3=max(SIG3[it],maxsig3);	

}	

if(ite%NBENREG==0){ 
cout<<"Maxsig11:"<<maxsig11<<endl;
cout<<"Minsig11:"<<minsig11<<endl;
cout<<"Maxsig12:"<<maxsig12<<endl;
cout<<"Minsig12:"<<minsig12<<endl;
cout<<"Maxsig13:"<<maxsig13<<endl;
cout<<"Minsig13:"<<minsig13<<endl;
cout<<"Maxsig22:"<<maxsig22<<endl;
cout<<"Minsig22:"<<minsig22<<endl;
cout<<"Maxsig23:"<<maxsig23<<endl;
cout<<"Minsig23:"<<minsig23<<endl;
cout<<"Maxsig33:"<<maxsig33<<endl;
cout<<"Minsig33:"<<minsig33<<endl;
cout<<"Maxsig1:"<<maxsig1<<endl;
cout<<"Minsig1:"<<minsig1<<endl;
cout<<"Maxsig2:"<<maxsig2<<endl;
cout<<"Minsig2:"<<minsig2<<endl;
cout<<"Maxsig3:"<<maxsig3<<endl;
cout<<"Minsig3:"<<minsig3<<endl;
cout<<"Maxvm:"<<maxvm<<endl;
cout<<"Minvm:"<<minvm<<endl;
cout<<"Maxtrac:"<<maxtrac<<endl;
cout<<"Mintrac:"<<mintrac<<endl;
}



}
